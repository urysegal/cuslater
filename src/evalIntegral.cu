#include "hip/hip_runtime.h"
//
// Created by gkluhana on 26/03/24.
//
// editted by MarkEwert03 on 13/05/24

#include <thrust/device_vector.h>

#include "../include/evalIntegral.h"

const double pi = 3.14159265358979323846;
#include <thread>
#define THREADS_PER_BLOCK 128
__constant__ float d_c[12];
__constant__ float d_alpha[4];
__constant__ float d_x_grid[600];
__constant__ float d_x_weights[600];

namespace cuslater {
double evaluateFourCenterIntegral(float *c, float *alpha, int nr, int nl, int nx, int ny, int nz,
                                  const std::string x1_type, double tol) {
    // read r grid
    std::cout << "Reading r Grid Files" << std::endl;
    const std::string r_filepath = "grid_files/r_" + std::to_string(nr) + ".grid";
    std::vector<float> r_nodes;
    std::vector<float> r_weights;
    read_r_grid_from_file(r_filepath, r_nodes, r_weights);

    // read l grid
    std::cout << "Reading l Grid Files" << std::endl;
    const std::string l_filepath = "grid_files/l_" + std::to_string(nl) + ".grid";
    std::vector<float> l_nodes_x;
    std::vector<float> l_nodes_y;
    std::vector<float> l_nodes_z;
    std::vector<float> l_weights;
    read_l_grid_from_file(l_filepath, l_nodes_x, l_nodes_y, l_nodes_z, l_weights);

    // Read x1 grid
    std::cout << "Reading x1 Grid Files" << std::endl;
    const std::string x1_filepath =
        "grid_files/x1_" + x1_type + "_1d_" + std::to_string(nx) + ".grid";
    std::vector<float> x1_nodes;
    std::vector<float> x1_weights;
    float a;
    float b;
    read_x1_1d_grid_from_file(x1_filepath, a, b, x1_nodes, x1_weights);

    // Initializing Device Variables
    std::cout << "Initializing Device Variables" << std::endl;
    unsigned int PX = x1_nodes.size();
    int threads = THREADS_PER_BLOCK;                // Max threads per block
    int blocks = (PX * PX + threads - 1) / threads; // Max blocks, better if multiple of SM = 80
    std::cout << "Total Threads: " << blocks * threads << std::endl;
    std::cout << "Total Grid Points: " << nx * ny * nz << std::endl;

    // cuda memory initialization
    hipMemcpyToSymbol(HIP_SYMBOL(d_c), c, 12 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), alpha, 4 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_x_grid), x1_nodes.data(), PX * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_x_weights), x1_weights.data(), PX * sizeof(float));

    // thrust devices
    thrust::device_vector<float> d_r_weights(nr);
    thrust::device_vector<float> d_l_weights(nl);
    thrust::device_vector<double> d_result(PX * PX);

    double *d_sum;
    HANDLE_CUDA_ERROR(hipMalloc(&d_sum, sizeof(double)));
    HANDLE_CUDA_ERROR(hipMemset(d_sum, 0, sizeof(double)));

    double sum = 0.0;
    double delta_sum = 0.0;
    int r_skipped = 0;

    // main loop
    std::cout << "Evaluating Integral for all values of r and l with\n";
    std::cout << "  a1=" << alpha[0] << ", a2=" << alpha[1] << ", a3=" << alpha[2]
              << ", a4=" << alpha[3] << "\n";
    std::cout << "  c1 = (" << c[0] << ", " << c[1] << ", " << c[2] << ")\n";
    std::cout << "  c2 = (" << c[3] << ", " << c[4] << ", " << c[5] << ")\n";
    std::cout << "  c3 = (" << c[6] << ", " << c[7] << ", " << c[8] << ")\n";
    std::cout << "  c4 = (" << c[9] << ", " << c[10] << ", " << c[11] << ")\n";
    std::cout << "  Tolerance = " << tol << "\n";

    for (int j = 0; j < nl; ++j) {
        for (int i = 0; i < nr; ++i) {
            delta_sum =
                evaluateInnerSum(nx, ny, nz, r_nodes[i], l_nodes_x[j], l_nodes_y[j], l_nodes_z[j],
                                 r_weights[i], l_weights[j], d_result, d_sum, blocks, threads, 0);
            if (delta_sum < tol) {
                r_skipped += nr - i;
                break;
            }
        }
        if (j % 50 == 0) {
            std::cout << "computed for l_j:" << j << "/" << nl << std::endl;
        }
    }
    HANDLE_CUDA_ERROR(hipMemcpy(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost));
    sum = sum * (4.0 / pi) * std::pow(alpha[0] * alpha[1] * alpha[2] * alpha[3], 1.5);

    // sum up result, multiply with constant and return
    std::cout << "Total values of r skipped for different l's: " << r_skipped << "/" << nr * nl
              << std::endl;
    return sum;
}

double evaluateInnerSum(unsigned int nx, unsigned int ny, unsigned int nz, float r, float l_x,
                        float l_y, float l_z, float r_weight, float l_weight,
                        thrust::device_vector<double> &__restrict__ d_result,
                        double *__restrict__ d_sum, int blocks, int threads, int gpu_num) {
    HANDLE_CUDA_ERROR(hipSetDevice(gpu_num));

    evaluateIntegrandReduceZ<<<blocks, threads>>>(nx, ny, nz, r, l_x, l_y, l_z,
                                                  raw_pointer_cast(d_result.data()));
    // Reduce vector on GPU within each block
    double delta_sum =
        thrust::reduce(d_result.begin(), d_result.end(), (double)0.0, thrust::plus<double>());
    // Accumulate result on device
    accumulateSum<<<1, 1>>>(delta_sum, r_weight, l_weight, d_sum);
    return delta_sum;
} // evaluateInner

__global__ void evaluateIntegrandReduceZ(int nx, int ny, int nz, float r, float l_x, float l_y,
                                         float l_z, double *__restrict__ res) {
    // gets index for current thread and blcok
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < nx * ny) {
        int y_idx = idx / nx;
        int x_idx = idx % nx;
        float xvalue = d_x_grid[x_idx];
        float yvalue = d_x_grid[y_idx];

        float wxy = d_x_weights[x_idx] * d_x_weights[y_idx];
        // compute function value
        // exp(-α1|x1-c1| - α2|x1-c2| - α3|x1-c3+r*l| - α4|x1-c4+r*l|)
        // note |a-b| = sqrt( (a.x-b.x)^2 + (a.y-b.y)^2 + (a.z-b.z)^2 )

        float xdiffc_1 = xvalue - d_c[0];            // x1.x - c1.x
        float ydiffc_1 = yvalue - d_c[1];            // x1.y - c1.y
        float xdiffc_2 = xvalue - d_c[3];            // x1.x - c2.x
        float ydiffc_2 = yvalue - d_c[4];            // x1.y - c2.y
        float xdiffc_3 = xvalue - d_c[6] + r * l_x;  // x1.x - c3.x + lx
        float ydiffc_3 = yvalue - d_c[7] + r * l_y;  // x1.y - c3.y + ly
        float xdiffc_4 = xvalue - d_c[9] + r * l_x;  // x1.x - c4.x + lx
        float ydiffc_4 = yvalue - d_c[10] + r * l_y; // x1.y - c4.y + ly

        // (x1.x - c1.x)^2 + (x1.y - c1.y)^2
        float xysq1 = xdiffc_1 * xdiffc_1 + ydiffc_1 * ydiffc_1;
        // (x1.x - c2.x)^2 + (x1.y - c2.y)^2
        float xysq2 = xdiffc_2 * xdiffc_2 + ydiffc_2 * ydiffc_2;
        // (x1.x - c3.x + lx)^2 + (x1.y - c3.y + ly)^2
        float xysq3 = xdiffc_3 * xdiffc_3 + ydiffc_3 * ydiffc_3;
        // (x1.x - c4.x + lx)^2 + (x1.y - c4.y + ly)^2
        float xysq4 = xdiffc_4 * xdiffc_4 + ydiffc_4 * ydiffc_4;

        double v = 0.0;

        for (int z_idx = 0; z_idx < nz; ++z_idx) {
            float zvalue = d_x_grid[z_idx];
            float wz = d_x_weights[z_idx];
            float zdiffc_1 = zvalue - d_c[2];                             // x1.z - c1.z
            float zdiffc_2 = zvalue - d_c[5];                             // x1.z - c2.z
            float zdiffc_3 = zvalue - d_c[8] + r * l_z;                   // x1.z - c3.z + lz
            float zdiffc_4 = zvalue - d_c[11] + r * l_z;                  // x1.z - c4.z + lz
            float term1 = d_alpha[0] * sqrt(xysq1 + zdiffc_1 * zdiffc_1); // α1 * ✓|x - c1|
            float term2 = d_alpha[1] * sqrt(xysq2 + zdiffc_2 * zdiffc_2); // α2 * ✓|x - c2|
            float term3 = d_alpha[2] * sqrt(xysq3 + zdiffc_3 * zdiffc_3); // α3 * ✓|x - c3 + r*l|
            float term4 = d_alpha[3] * sqrt(xysq4 + zdiffc_4 * zdiffc_4); // α4 * ✓|x - c4 + r*l|
            float exponent = -term1 - term2 - term3 - term4 + r;
            v += exp(exponent) * wxy * wz;
        }
        res[idx] = v;
    }
} // evaluateReduceInnerIntegrandz

__global__ void accumulateSum(double result, float r_weight, float l_weight,
                              double *__restrict__ d_sum) {
    atomicAdd(d_sum, result * r_weight * l_weight);
}

} // namespace cuslater
