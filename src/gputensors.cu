/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <array>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <iostream>
#include <string>
#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>
#include "../include/cuslater.cuh"
#include <hip/hip_runtime.h>

namespace cuslater {

void
init_cuslater()
{
    hipError_t res;
    if ( (res=hipInit(0)) != hipSuccess ) {
        fprintf(stderr, "Cannot init CUDA: %d\n",res);
        exit(1);
    }
}


int hadamard(std::vector<int> &modes, std::unordered_map<int, int64_t> &extent, const double *A, const double *C,
             double *D)
{

    hipDataType typeA = HIP_R_64F;
    hipDataType typeC = HIP_R_64F;
    hipDataType typeCompute = HIP_R_64F;

    double alpha = 1;
    double gamma = 1;

    /**********************
     * Computing: D_{a,b,c} =  A_{a,b,c}  *  C_{a,b,c}
     **********************/

    //std::vector<int> modeC{'a','b','c'};
    //std::vector<int> modeA{'c','b','a'};
    int nmodes = modes.size();
//    int nmodeC = modeC.size();


    //extent['a'] = 400;
    //extent['b'] = 200;
    //extent['c'] = 300;

    std::vector<int64_t> extentA;
    for (auto mode : modes)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentC;
    for (auto mode : modes)
        extentC.push_back(extent[mode]);
    std::vector<int64_t> extentD;
    for (auto mode : modes)
        extentD.push_back(extent[mode]);


    /**********************
     * Allocating data
     **********************/

    size_t elements = 1;
    for (auto mode : modes)
        elements *= extent[mode];

    size_t sizeM = sizeof(double) * elements;
    printf("Total memory: %.2f GiB\n",(sizeM + sizeM)/1024./1024./1024);

    void *A_d, *C_d, *D_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**) &A_d, sizeM));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &C_d, sizeM));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &D_d, sizeM));

    /*******************
     * Initialize data
     *******************/

    hipDeviceSynchronize();
    GPUTimer timer;
    timer.start();

    HANDLE_CUDA_ERROR(hipMemcpy2DAsync(C_d, sizeM, C, sizeM, sizeM, 1, hipMemcpyDefault, 0));
    HANDLE_CUDA_ERROR(hipMemcpy2DAsync(A_d, sizeM, A, sizeM, sizeM, 1, hipMemcpyDefault, 0));

    /*************************
     * Memcpy perf
     *************************/

    double minTimeMEMCPY = 1e100;
    hipDeviceSynchronize();
    minTimeMEMCPY = timer.seconds();

    /*************************
     * cuTENSOR
     *************************/
    hiptensorStatus_t err;
    hiptensorHandle_t handle;
    HANDLE_TENSOR_ERROR(cutensorInit(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/
    hiptensorTensorDescriptor_t descA;
    HANDLE_TENSOR_ERROR(hiptensorInitTensorDescriptor( &handle,
                                               &descA,
                                               nmodes,
                                               extentA.data(),
                                               NULL /* stride */,
                                               typeA, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descC;
    HANDLE_TENSOR_ERROR(hiptensorInitTensorDescriptor( &handle,
                                               &descC,
                                               nmodes,
                                               extentC.data(),
                                               NULL /* stride */,
                                               typeC, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descD;
    HANDLE_TENSOR_ERROR(hiptensorInitTensorDescriptor( &handle,
                                               &descD,
                                               nmodes,
                                               extentD.data(),
                                               NULL /* stride */,
                                               typeC, HIPTENSOR_OP_IDENTITY));


        HANDLE_CUDA_ERROR(hipDeviceSynchronize());
        timer.start();
        err = cutensorElementwiseBinary(&handle,
                                        (void*)&alpha, A_d, &descA, modes.data(),
                                        (void*)&gamma, C_d, &descC, modes.data(),
                                        D_d, &descD, modes.data(),
                                        HIPTENSOR_OP_MUL, typeCompute, 0 /* stream */);
        auto time = timer.seconds();
        if (err != HIPTENSOR_STATUS_SUCCESS)
        {
            printf("ERROR: %s\n", hiptensorGetErrorString(err) );
        }

    HANDLE_CUDA_ERROR(hipMemcpy2D(D_d, sizeM, D_d, sizeM, sizeM, 1, hipMemcpyDefault));

    /*************************/


    double transferedBytes = sizeM;
    transferedBytes += ((float)alpha != 0.f) ? sizeM : 0;
    transferedBytes += ((float)gamma != 0.f) ? sizeM : 0;
    transferedBytes /= 1e9;
    printf("cuTensor: %.2f GB/s\n", transferedBytes / time);
    printf("memcpy: %.2f GB/s\n", 2 * sizeM / minTimeMEMCPY / 1e9 );

    if (A_d) hipFree(A_d);
    if (C_d) hipFree(C_d);
    if (D_d) hipFree(D_d);

    return 0;
}




}
